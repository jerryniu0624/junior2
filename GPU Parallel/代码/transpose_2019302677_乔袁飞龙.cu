#include "hip/hip_runtime.h"
#include <stdio.h>
#include<time.h>
#include<math.h>
#include <hip/hip_runtime.h>


#define TILE_DIM    32
#define BLOCK_ROWS  32

// This sample assumes that MATRIX_SIZE_X = MATRIX_SIZE_Y
int MATRIX_SIZE_X = 1024;
int MATRIX_SIZE_Y = 1024;

#define IDX(r,c) ((r)*width+(c))
#define AIDX(r,c) ((r)*height+(c))
#define NUM_REPS  10


// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------
__global__ void copy(float *odata, float *idata, int width, int height){
	int xIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			odata[IDX(yIdx+offset,xIdx)]=idata[IDX(yIdx+offset,xIdx)];
	}
}

__global__ void copySharedMem(float *odata, float *idata, int width, int height)
{
	__shared__ float temp[TILE_DIM][TILE_DIM];
	int xIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	int tmpx = threadIdx.x;
	int tmpy = threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			temp[tmpy+offset][tmpx]=idata[IDX(yIdx+offset,xIdx)];
	}
	__syncthreads();
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			odata[IDX(yIdx+offset,xIdx)]=temp[tmpy+offset][tmpx];
	}
}

// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float *idata, int width, int height){
	int yIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int xIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			odata[AIDX(xIdx,yIdx+offset)]=idata[IDX(yIdx+offset,xIdx)];//合并写，非合并读
	}
}

// coalesced transpose (with bank conflicts)
__global__ void transposeCoalesced(float *odata, float *idata, int width, int height)
{	
	__shared__ float temp[TILE_DIM][TILE_DIM];
	int xIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	int tmpx = threadIdx.x;
	int tmpy = threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			temp[tmpy+offset][tmpx]=idata[IDX(yIdx+offset,xIdx)];//合并读
	}
	__syncthreads();
	xIdx = blockIdx.y * TILE_DIM + threadIdx.x;
	yIdx = blockIdx.x * TILE_DIM + threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<height&&yIdx+offset<width)
			odata[AIDX(yIdx+offset,xIdx)]=temp[tmpx][tmpy+offset];//合并写
	}
}
__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height)
{
	__shared__ float temp[TILE_DIM][TILE_DIM+1];
	int xIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	int tmpx = threadIdx.x;
	int tmpy = threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<width&&yIdx+offset<height)
			temp[tmpy+offset][tmpx]=idata[IDX(yIdx+offset,xIdx)];
	}
	__syncthreads();
	xIdx = blockIdx.y * TILE_DIM + threadIdx.x;
	yIdx = blockIdx.x * TILE_DIM + threadIdx.y;
	for(int offset=0;offset<TILE_DIM;offset+=BLOCK_ROWS){
		if(xIdx<height&&yIdx+offset<width)
			odata[AIDX(yIdx+offset,xIdx)]=temp[tmpx][tmpy+offset];
	}
}

void computeTransposeGold(float *gold, float *idata,const  int size_x, const  int size_y){
	for (int y = 0; y < size_y; ++y)
		for (int x = 0; x < size_x; ++x)
			gold[(x * size_y) + y] = idata[(y * size_x) + x];
}


bool compareData(float* a, float* b, int n) {
	for (int i = 0; i < n; i++)
		if (abs(a[i] - b[i]) > 0.0001)
			return false;
	return true;
}


int main(){	
	int size_x = MATRIX_SIZE_X;
	int size_y = MATRIX_SIZE_Y;
	// CUDA events
	hipEvent_t start, stop; hipEventCreate(&start);	hipEventCreate(&stop); float outerTime;
	// size of memory required to store the matrix
	const  int mem_size = sizeof(float) * MATRIX_SIZE_X*MATRIX_SIZE_Y;
	float *idata, *odata, *transposeGold, *gold;
	hipMallocManaged(&idata, mem_size);
	hipMallocManaged(&odata, mem_size);
	hipMallocManaged(&transposeGold, mem_size);
	hipMallocManaged(&gold, mem_size);



	// step1: initalize host data
	for (int i = 0; i < (MATRIX_SIZE_X*MATRIX_SIZE_Y); ++i)
		idata[i] = (float)i;

	// step2: Compute reference transpose solution
	clock_t start_t = clock();
	for (int i = 0; i < NUM_REPS; i++)
		computeTransposeGold(transposeGold, idata, MATRIX_SIZE_X, MATRIX_SIZE_Y);
	clock_t finish_t = clock();
	float total_t = (float)(finish_t - start_t) / CLOCKS_PER_SEC*1000; 
	printf("Transposed: CPU Elapsed time:%.6f ms.\n", total_t);


	// execution configuration parameters
	dim3 grid(size_x / TILE_DIM+1, size_y / TILE_DIM+1), threads(TILE_DIM, BLOCK_ROWS);
	//step3:  warmup to avoid timing startup
	copy<<<grid,threads>>>(odata,idata,size_x,size_y);
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++)
		copy <<<grid, threads >>>(odata, idata, size_x, size_y);
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	
	hipEventElapsedTime(&outerTime, start, stop);
	printf("Simple Copy: GPU Elapsed time:%.6f ms.\n", outerTime);

	//Step4: copySharedMem
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++)
		copySharedMem << <grid, threads >> >(odata, idata, size_x, size_y);
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	
	hipEventElapsedTime(&outerTime, start, stop);
	printf("SharedMem Copy: GPU Elapsed time:%.6f ms.\n", outerTime);

	//Step5: transposeNaive
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++)
		transposeNaive <<<grid, threads >>>(odata, idata, size_x, size_y);
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);
	hipEventElapsedTime(&outerTime, start, stop);
	if (compareData(transposeGold, odata, size_x*size_y))
		printf("TransposeNaive (True): GPU Elapsed time:%.6f ms.\n", outerTime);

	//Step6: transposeCoalesced
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++)
		transposeCoalesced <<<grid, threads >> >(odata, idata, size_x, size_y);
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);
	hipEventElapsedTime(&outerTime, start, stop);
	if (compareData(transposeGold, odata, size_x*size_y))
		printf("TransposeCoalesced(True): GPU Elapsed time:%.6f ms.\n", outerTime);


	//Step7: transposeNoBankConflicts
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++)
		transposeNoBankConflicts << <grid, threads >> >(odata, idata, size_x, size_y);
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);
	hipEventElapsedTime(&outerTime, start, stop);
	if (compareData(transposeGold, odata, size_x*size_y))
		printf("TransposeNoBankConflicts(True): GPU Elapsed time:%.6f ms.\n", outerTime);
	hipFree(idata); hipFree(odata); hipFree(transposeGold); hipFree(gold);
	return 1;
}