#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include<time.h>

#define BLKDIM 1024
#define N_OF_BLOCKS   1024
/* N must be an integer multiple of BLKDIM */
#define N ((N_OF_BLOCKS)*(BLKDIM))

#define NUM_REPS  100

int reduceCPU(int *data, int size)
{
	int sum = data[0];
	for (int i = 1; i < size; i++)
		sum += data[i];
	return sum;
}


__global__ void reduce1(int *g_idata, int *g_odata, int n) {
	__shared__ int sdata[BLKDIM];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			sdata[tid] += sdata[tid + s];
	}
			__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2(int *g_idata, int *g_odata, int n) {
	__shared__ int sdata[BLKDIM];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// __global__ void reduce0( int *a, int* sums, int n )
// {
//    __shared__ int temp[BLKDIM];
//    temp[threadIdx.x] = a[threadIdx.x + blockIdx.x * blockDim.x];
//    __syncthreads(); 
//    if ( 0 == threadIdx.x) {
//        int i, my_sum = 0;
//        for (i=0; i<blockDim.x; i++)
// 	    {my_sum += temp[i];}
//        sums[blockIdx.x] = my_sum;
//    }
// }

//__global__ void reduce1(int *a, int* sums, int n)
//{
//	__shared__ int temp[BLKDIM];
//	int lindex = threadIdx.x;
//	int bindex = blockIdx.x;
//	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
//	int bsize = blockDim.x / 2;
//
//	temp[lindex] = a[gindex];  __syncthreads();
//
//	/* All threads within the block cooperate to compute the local sum */
//	while (bsize > 0) {
//		if (lindex < bsize) {temp[lindex] += temp[lindex + bsize];}
//		bsize = bsize / 2;	
//		__syncthreads();
//	}
//	if (0 == lindex) {sums[bindex] = temp[0];}
//}

__global__ void reduce3(int *g_idata, int *g_odata, unsigned int n)
{
	__shared__ int sdata[BLKDIM];

	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = g_idata[i];
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce4(int *g_idata, int *g_odata, unsigned int n)
{
	__shared__ int sdata[BLKDIM];

	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	sdata[tid] = 0;
	if ( i < n ) 
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


__global__ void reduce6( int* g_idata,  int* g_odata,  int n) {
	__shared__ int sdata[BLKDIM];
	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	sdata[tid] = 0;
	if (i < n)
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	__syncthreads();

	if (tid < 512) 
		sdata[tid] += sdata[tid + 512]; __syncthreads();
	if(tid < 256)
		sdata[tid] += sdata[tid + 256]; __syncthreads();
	if (tid < 128)
		sdata[tid] += sdata[tid + 128]; __syncthreads();
	if (tid < 64)
		sdata[tid] += sdata[tid + 64];  __syncthreads();

	//	write result for this block to global mem
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32]; __syncthreads();
		sdata[tid] += sdata[tid + 16]; __syncthreads();
		sdata[tid] += sdata[tid + 8]; __syncthreads();
		sdata[tid] += sdata[tid + 4]; __syncthreads();
		sdata[tid] += sdata[tid + 2]; __syncthreads();
		sdata[tid] += sdata[tid + 1]; __syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce5(int *g_idata, int *g_odata, int n)
{

	__shared__ int sdata[BLKDIM];
	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	sdata[tid] = 0;
	if ( i < n ) 
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>32; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	//	write result for this block to global mem
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32]; __syncthreads();
		sdata[tid] += sdata[tid + 16]; __syncthreads();
		sdata[tid] += sdata[tid + 8]; __syncthreads();
		sdata[tid] += sdata[tid + 4]; __syncthreads();
		sdata[tid] += sdata[tid + 2]; __syncthreads();
		sdata[tid] += sdata[tid + 1]; __syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0)
	    g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce51(int *g_idata, int *g_odata, int n)
{

	__shared__ int sdata[BLKDIM];
	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>32; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	//	write result for this block to global mem
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32]; __syncthreads();
		sdata[tid] += sdata[tid + 16]; __syncthreads();
		sdata[tid] += sdata[tid + 8]; __syncthreads();
		sdata[tid] += sdata[tid + 4]; __syncthreads();
		sdata[tid] += sdata[tid + 2]; __syncthreads();
		sdata[tid] += sdata[tid + 1]; __syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}



int main( void ) 
{

	// step0: CUDA events
	hipEvent_t start, stop; hipEventCreate(&start);	hipEventCreate(&stop); float outerTime;

	// step0: malloc space
	int *a, *b; int i, s=0;
    hipMallocManaged(&a,N*sizeof(int));    hipMallocManaged(&b,N_OF_BLOCKS *sizeof(int));
    
	// step1: initialization
	for (i = 0; i < N; i++) { a[i] = 2; }
	
	hipMemset(&b, 0, N_OF_BLOCKS);

	clock_t start_t = clock();
	for (int i = 0; i < NUM_REPS; i++)
		s = reduceCPU(a,N);
	clock_t finish_t = clock();
	outerTime = (float)(finish_t - start_t) / CLOCKS_PER_SEC * 1000;
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction with CPU:  Elapsed time:%.6f ms.\n", outerTime); }



	// warm up 
	reduce1 <<<N_OF_BLOCKS, BLKDIM >>>(a, b, N);

	// step2: running  sum0
	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce1 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N); 
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	for (i = 0; i < N_OF_BLOCKS; i++) { s += b[i]; }
    if ( s != 2*N ) {printf("Check FAILED: Expected %d, got %d\n", 2*N, s);    } 
	else {  printf("Reduction1: GPU Elapsed time:%.6f ms.\n", outerTime); }

	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce2 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N); 
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	for (i = 0; i < N_OF_BLOCKS; i++) { s += b[i]; }
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction2: GPU Elapsed time:%.6f ms.\n", outerTime); }
 

	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce3 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	for (i = 0; i < N_OF_BLOCKS; i++) { s += b[i]; }
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction3: GPU Elapsed time:%.6f ms.\n", outerTime); }


	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce4 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N);
		hipDeviceSynchronize();		
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	for (i = 0; i < N_OF_BLOCKS; i++) { s += b[i]; }
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction4: GPU Elapsed time:%.6f ms.\n", outerTime); }


	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce5<<<N_OF_BLOCKS, BLKDIM >> > (a, b, N);
		hipDeviceSynchronize();	
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	
	for (i = 0; i < N_OF_BLOCKS; i++) { 
		s += b[i];
	}
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction5: GPU Elapsed time:%.6f ms.\n", outerTime); }



	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce6 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	
	for (i = 0; i < N_OF_BLOCKS; i++) {
		s += b[i];
	}
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction5-1: GPU Elapsed time:%.6f ms.\n", outerTime); }

	hipEventRecord(start, 0);
	for (int i = 0; i < NUM_REPS; i++) {
		reduce6 << <N_OF_BLOCKS, BLKDIM >> > (a, b, N);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&outerTime, start, stop);
	s = 0; 	for (i = 0; i < N_OF_BLOCKS; i++) { s += b[i]; }
	if (s != 2 * N) { printf("Check FAILED: Expected %d, got %d\n", 2 * N, s); }
	else { printf("Reduction6: GPU Elapsed time:%.6f ms.\n", outerTime); }

	// step4: free
	hipFree(a); 	hipFree(b);


    return 0;
}
